#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include "gpuhash.h"
#include "sha1.h"

#define BUFLEN (500*1024*1024)
#define PACKET_SIZE (4*1024)
#define BLOCK_SIZE 128

using namespace std;

__constant__ int thread_counts;

__device__ unsigned int roll(const unsigned int value,
                const unsigned int steps)
        {
            return ((value << steps) | (value >> (32 - steps)));
        }


__device__ void clearwBuffert(unsigned int* buffert)
        {
            for (int pos = 16; --pos >= 0;)
            {
                buffert[pos] = 0;
            }
        }

__device__ void innerhash(unsigned int* result, unsigned int* w)
        {
            unsigned int a = result[0];
            unsigned int b = result[1];
            unsigned int c = result[2];
            unsigned int d = result[3];
            unsigned int e = result[4];

            int round = 0;

            #define sha1macro(func,val) \
            { \
                const unsigned int t = roll(a, 5) + (func) + e + val + w[round]; \
                e = d; \
                d = c; \
                c = roll(b, 30); \
                b = a; \
                a = t; \
            }

            while (round < 16)
            {
                sha1macro((b & c) | (~b & d), 0x5a827999)
                ++round;
            }
            while (round < 20)
            {
                w[round] = roll((w[round - 3] ^ w[round - 8] ^ w[round - 14] ^ w[round - 16]), 1);
                sha1macro((b & c) | (~b & d), 0x5a827999)
                ++round;
            }
            while (round < 40)
            {
                w[round] = roll((w[round - 3] ^ w[round - 8] ^ w[round - 14] ^ w[round - 16]), 1);
                sha1macro(b ^ c ^ d, 0x6ed9eba1)
                ++round;
            }
            while (round < 60)
            {
                w[round] = roll((w[round - 3] ^ w[round - 8] ^ w[round - 14] ^ w[round - 16]), 1);
                sha1macro((b & c) | (b & d) | (c & d), 0x8f1bbcdc)
                ++round;
            }
            while (round < 80)
            {
                w[round] = roll((w[round - 3] ^ w[round - 8] ^ w[round - 14] ^ w[round - 16]), 1);
                sha1macro(b ^ c ^ d, 0xca62c1d6)
                ++round;
            }

            #undef sha1macro

            result[0] += a;
            result[1] += b;
            result[2] += c;
            result[3] += d;
            result[4] += e;
        }





__device__ void calculate(const void* src, const int bytelength, unsigned char* hash)
    {
        // Init the result array.
        unsigned int result[5] = { 0x67452301, 0xefcdab89, 0x98badcfe, 0x10325476, 0xc3d2e1f0 };

        // Cast the void src pointer to be the byte array we can work with.
        const unsigned char* sarray = (const unsigned char*) src;

        // The reusable round buffer
        unsigned int w[80];

        // Loop through all complete 64byte blocks.
        const int endOfFullBlocks = bytelength - 64;
        int endCurrentBlock;
        int currentBlock = 0;

        while (currentBlock <= endOfFullBlocks)
        {
            endCurrentBlock = currentBlock + 64;

            // Init the round buffer with the 64 byte block data.
            for (int roundPos = 0; currentBlock < endCurrentBlock; currentBlock += 4)
            {
                // This line will swap endian on big endian and keep endian on little endian.
                w[roundPos++] = (unsigned int) sarray[currentBlock + 3]
                        | (((unsigned int) sarray[currentBlock + 2]) << 8)
                        | (((unsigned int) sarray[currentBlock + 1]) << 16)
                        | (((unsigned int) sarray[currentBlock]) << 24);
            }
            innerhash(result, w);
        }

        // Handle the last and not full 64 byte block if existing.
        endCurrentBlock = bytelength - currentBlock;
        clearwBuffert(w);
        int lastBlockBytes = 0;
        for (;lastBlockBytes < endCurrentBlock; ++lastBlockBytes)
        {
            w[lastBlockBytes >> 2] |= (unsigned int) sarray[lastBlockBytes + currentBlock] << ((3 - (lastBlockBytes & 3)) << 3);
        }
        w[lastBlockBytes >> 2] |= 0x80 << ((3 - (lastBlockBytes & 3)) << 3);
        if (endCurrentBlock >= 56)
        {
            innerhash(result, w);
            clearwBuffert(w);
        }
        w[15] = bytelength << 3;
        innerhash(result, w);

        for (int hashByte = 20; --hashByte >= 0;)
        {
            hash[hashByte] = (result[hashByte >> 2] >> (((3 - hashByte) & 0x3) << 3)) & 0xff;
        }
    }


__global__
void calculateHash(unsigned char *gdata, unsigned char *hash)
{

    int i=blockIdx.x * blockDim.x + threadIdx.x;
    if(i<thread_counts){
    calculate(&gdata[i*PACKET_SIZE],PACKET_SIZE,&hash[i*20]);
    }
}



int calc_gpu(unsigned char *data1, int charRead, thread_data_t * &t)
{

        unsigned char *gdata;
        unsigned char *hash;
        const int gdsize = charRead*sizeof(char);
        
        hipMalloc((void**)&gdata,gdsize);
        hipError_t cuda_result_code = hipGetLastError();
        
        if (cuda_result_code!=hipSuccess) {
           printf("gdata message: %s\n",hipGetErrorString(cuda_result_code));
        }

        int threads = charRead/(PACKET_SIZE);
        
	    int blocks = threads/BLOCK_SIZE;
	    if(threads%BLOCK_SIZE) 
		      blocks++;
        
        const int hash_size = 20 * blocks * BLOCK_SIZE * sizeof(unsigned char);
        
        hipMalloc((void**)&hash, hash_size);

        cuda_result_code = hipGetLastError();

        if (cuda_result_code!=hipSuccess) {
                  printf("hash message: %s\n",hipGetErrorString(cuda_result_code));
               }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Start timer
        hipEventRecord(start);

        hipMemcpy( gdata, data1, gdsize, hipMemcpyHostToDevice );

        if (cuda_result_code!=hipSuccess) {
                   printf(" memcpy message: %s\n",hipGetErrorString(cuda_result_code));
                }
                
        hipMemcpyToSymbol(HIP_SYMBOL(thread_counts), &threads, sizeof(int));

        calculateHash<<<blocks, BLOCK_SIZE>>>(gdata,hash);
        
        hipDeviceSynchronize();
        
        if (cuda_result_code!=hipSuccess) {
                   printf(" hashcalucation message: %s\n",hipGetErrorString(cuda_result_code));
                }

        unsigned char * hash_host = new unsigned char[hash_size];
        hipMemcpy( hash_host, hash, hash_size, hipMemcpyDeviceToHost);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        cuda_result_code = hipGetLastError();

        if (cuda_result_code!=hipSuccess) {
                  printf("hash message: %s\n",hipGetErrorString(cuda_result_code));
               }

        cout <<"\nTime taken : "<<milliseconds<<"ms"<<endl;
        hipFree( gdata );
        hipFree( hash );

        cout<< charRead;
        int remainders = charRead % PACKET_SIZE;
        if (remainders)
            threads++;
        t = new thread_data_t[threads];
        int len=PACKET_SIZE;
        for(int i=0;i<threads;i++)
        {
            if(i==threads-1){
                len = remainders;
                sha1::calc(&data1[i*PACKET_SIZE],len,&hash_host[i*20]);
                cout<<"len :"<<len<<endl;
            }
            t[i].data = new char[len];
            memcpy ( t[i].data, &data1[i*PACKET_SIZE], len);
            t[i].len = len;
            memcpy ( t[i].hash, &hash_host[i*20], 20);
            /*
            cout<<"Hash :";
        	for(int j=0;j<20;j++)
        		printf("%02x",hash_host[i*20+j]);
            cout<<endl; */
        }

        delete[] hash_host;
    printf("Completed");
    return threads;
}
